
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>

__global__ void iteration_inner(double *** u, double *** uold, double *** f, int N, double delta2, double frac) {
    int k = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int i = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if ((i < N + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] \
                        + uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);
    }
}



void iteration(double *** u, double *** uold, double *** f, int N) {
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;
    // Blocks and threads
    dim3 dimBlock(32,4,2);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,(N+dimBlock.z-1)/dimBlock.z);
    //dim3 dimGrid(((200+1)+dimBlock.x-1)/dimBlock.x,((200+1)+dimBlock.y-1)/dimBlock.y,((200+1)+dimBlock.z-1)/dimBlock.z);
    // Do iteration
    //double start = omp_get_wtime();
    iteration_inner<<<dimGrid, dimBlock>>>(u, uold, f, N, delta2, frac);
    hipDeviceSynchronize();
    //double end = omp_get_wtime() - start;
    //printf("Time = %f\n",end);
}