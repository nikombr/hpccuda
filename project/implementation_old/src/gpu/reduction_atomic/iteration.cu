
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>

__global__ void iteration_inner(double *** u, double *** uold, double *** f, int N, double *res, double delta2, double frac) {
    double val = 0;
    int k = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int i = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if ((i < N + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] \
                        + uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);
        val = u[i][j][k] - uold[i][j][k];
    }

    double reg = val*val;
    atomicAdd(res,reg);
    
}

__global__ void init_zero(double *res) {
    *res = 0.0;
}

void iteration(double *** u, double *** uold, double *** f, int N, double *sum) {
    init_zero<<<1, 1>>>(sum);
    hipDeviceSynchronize();
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;
    // Blocks and threads
    dim3 dimBlock(32,4,2);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,(N+dimBlock.z-1)/dimBlock.z);

    // Do iteration
    iteration_inner<<<dimGrid, dimBlock>>>(u, uold, f, N, sum, delta2, frac);
    hipDeviceSynchronize();
}