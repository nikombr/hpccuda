
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>


__global__ void iteration_inner(double *** u, double *** uold, double *** f, int N, int width, double delta2, double frac) {
    
    int k = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int i = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if ((i < width + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] \
                        + uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);
    }
}

void iteration(double *** u, double *** uold, double *** f, int N, int width) {
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;
    // Blocks and threads
    dim3 dimBlock(32,4,2);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,(width+dimBlock.z-1)/dimBlock.z);
    // Kernel call
    iteration_inner<<<dimGrid, dimBlock>>>(u, uold, f, N, width, delta2, frac);
    hipDeviceSynchronize();
}